#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <cutil.h>

#define TPB 512

__global__ void
PolonaiseKernel(float *);

void runPolonaiseKernel(float *data, int numOfAgents) {
    int gpu_count;
    hipGetDeviceCount(&gpu_count);
    if (gpu_count < 1) {
        return;
    }
    
    hipSetDevice(0);
    
    const unsigned int mem_size = sizeof(float) * numOfAgents * 6;
    
    float* d_data;
    hipMalloc((void **) &d_data, mem_size);
    hipMemcpy(d_data, data, mem_size, hipMemcpyHostToDevice);
    
    dim3 grid(numOfAgents/TPB,1,1);
    dim3 threads(TPB,1,1);
    
    // call kernel
    PolonaiseKernel<<<grid, threads>>>(d_data);
    CUT_CHECK_ERROR("Kernel execution failed");
    
    hipDeviceSynchronize();
    
    hipMemcpy(data, d_data, mem_size, hipMemcpyDeviceToHost);
    
    hipFree(d_data);
}
