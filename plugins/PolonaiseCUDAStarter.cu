#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <cutil.h>
#include <stdio.h>
#include "OpenSteer/VehicleData.h"
#include "CUDAKernelOptions.cu"

__global__ void
findFollowerKernel(VehicleData *vehicleData, float3 *seekVectors);

__global__ void
steerForSeekKernel(VehicleData *vehicleData, float3 *seekVectors, float3 *steeringVectors, float weight, kernel_options options);

__global__ void
updateKernel(VehicleData *vehicleData, VehicleConst *vehicleConst, float3 *steeringVectors, float elapsedTime, kernel_options options);

static VehicleData* d_vehicleData = NULL;
static VehicleConst* d_vehicleConst = NULL;
static float3* d_steeringVectors = NULL;
static float3* d_seekVectors = NULL;

void runPolonaiseKernel(VehicleData *h_vehicleData, VehicleConst *h_vehicleConst, int numOfAgents, float elapsedTime) {

    int gpu_count;
    hipGetDeviceCount(&gpu_count);
    if (gpu_count < 1) {
        return;
    }
    
    hipSetDevice(0);
    
    dim3 grid(numOfAgents/TPB,1,1);
    dim3 threads(TPB,1,1);
    
    // prepare memory for steeringVectors
    const unsigned int mem_size_steering = sizeof(float3) * numOfAgents;
    if (d_steeringVectors == NULL) {
        hipMalloc((void **) &d_steeringVectors, mem_size_steering);
    }
    
    hipMemset(d_steeringVectors, 0, mem_size_steering);

    
    // prepare vehicle data
    const unsigned int mem_size_vehicle = sizeof(VehicleData);
    if (d_vehicleData == NULL) {
        hipMalloc((void **) &d_vehicleData, mem_size_vehicle);
        hipMemcpy(d_vehicleData, h_vehicleData, mem_size_vehicle, hipMemcpyHostToDevice);
    }
    
    const unsigned int mem_size_vehicle_const = sizeof(VehicleConst);
    if (d_vehicleConst == NULL) {
        hipMalloc((void **)&d_vehicleConst, mem_size_vehicle_const);
        hipMemcpy(d_vehicleConst, h_vehicleConst, mem_size_vehicle_const, hipMemcpyHostToDevice);
    }
    
    // prepare steerForSeekKernel
    if (d_seekVectors == NULL) {
        const unsigned int mem_size_seek_vectors = sizeof(float3) * numOfAgents;
        hipMalloc((void **) &d_seekVectors, mem_size_seek_vectors);
    }
    
    // create and start timer
//    unsigned int timer = 0;
//    CUT_SAFE_CALL(cutCreateTimer(&timer));
//    CUT_SAFE_CALL(cutStartTimer(timer));
    
    // call findFollowerKernel
    findFollowerKernel<<<grid, threads>>>(d_vehicleData, d_seekVectors);
    //CUT_CHECK_ERROR("Kernel execution failed");
    
    // stop and destroy timer
//    CUT_SAFE_CALL(cutStopTimer(timer));
//    printf("Raw processing time (findFollowerKernel): %f (ms) \n", cutGetTimerValue(timer));
//    CUT_SAFE_CALL(cutDeleteTimer(timer));
//    CUT_SAFE_CALL(cutCreateTimer(&timer));
//    CUT_SAFE_CALL(cutStartTimer(timer));

    // call steerForSeekKernel
    steerForSeekKernel<<<grid, threads>>>(d_vehicleData, d_seekVectors, d_steeringVectors, 1.f, NONE);
    //CUT_CHECK_ERROR("Kernel execution failed");
            
    // stop and destroy timer
//    CUT_SAFE_CALL(cutStopTimer(timer));
//    printf("Raw processing time (steerForSeekKernel): %f (ms) \n", cutGetTimerValue(timer));
//    CUT_SAFE_CALL(cutDeleteTimer(timer));
//    CUT_SAFE_CALL(cutCreateTimer(&timer));
//    CUT_SAFE_CALL(cutStartTimer(timer));

    // call updateKernel
    updateKernel<<<grid, threads>>>(d_vehicleData, d_vehicleConst, d_steeringVectors, elapsedTime, NONE);
    //CUT_CHECK_ERROR("Kernel execution failed");
    
    //hipDeviceSynchronize();
    
    // stop and destroy timer
//    CUT_SAFE_CALL(cutStopTimer(timer));
//    printf("Raw processing time (updateKernel): %f (ms) \n", cutGetTimerValue(timer));
//    CUT_SAFE_CALL(cutDeleteTimer(timer));
//    CUT_SAFE_CALL(cutCreateTimer(&timer));
//    CUT_SAFE_CALL(cutStartTimer(timer));
    
    hipMemcpy(h_vehicleData, d_vehicleData, mem_size_vehicle, hipMemcpyDeviceToHost);
    
    // stop and destroy timer
//    CUT_SAFE_CALL(cutStopTimer(timer));
//    printf("Memcpy time: %f (ms) \n", cutGetTimerValue(timer));
//    CUT_SAFE_CALL(cutDeleteTimer(timer));    
    
    
    //hipFree(vehicleData);
}

void endPolonaise(void)
{
    hipFree(d_vehicleData);
    hipFree(d_vehicleConst);
    hipFree(d_steeringVectors);
    hipFree(d_seekVectors);
    
    d_vehicleData = NULL;
    d_vehicleConst = NULL;
    d_steeringVectors = NULL;
    d_seekVectors = NULL;
}
