#include "hip/hip_runtime.h"
#ifndef _STEER_TO_AVOID_OBSTACLES_
#define _STEER_TO_AVOID_OBSTACLES_

#include <cutil.h>
#include "VehicleData.h"
#include "ObstacleData.h"
#include "CUDAFloatUtilities.cu"
#include "CUDAVectorUtilities.cu"
#include "CUDAKernelOptions.cu"

#define CHECK_BANK_CONFLICTS 0
#if CHECK_BANK_CONFLICTS
#define V_F(i) (CUT_BANK_CHECKER(((float*)velocity), i))
#define F_F(i) (CUT_BANK_CHECKER(((float*)forward), i))
#define P_F(i) (CUT_BANK_CHECKER(((float*)position), i))
#define S_F(i) (CUT_BANK_CHECKER(((float*)steering), i))
#define A_F(i) (CUT_BANK_CHECKER(((float*)avoidance), i))
#define LC_F(i) (CUT_BANK_CHECKER(((float*)localcenter), i))
#define V(i) (CUT_BANK_CHECKER(velocity, i))
#define F(i) (CUT_BANK_CHECKER(forward, i))
#define P(i) (CUT_BANK_CHECKER(position, i))
#define S(i) (CUT_BANK_CHECKER(steering, i))
#define SP(i) (CUT_BANK_CHECKER(speed, i))
#define A(i) (CUT_BANK_CHECKER(avoidance, i))
#define LC(i) (CUT_BANK_CHECKER(localcenter, i))
#else
#define V_F(i) ((float*)velocity)[i]
#define F_F(i) ((float*)forward)[i]
#define P_F(i) ((float*)position)[i]
#define S_F(i) ((float*)steering)[i]
#define A_F(i) ((float*)avoidance)[i]
#define LC_F(i) ((float*)localcenter)[i]
#define V(i) velocity[i]
#define F(i) forward[i]
#define P(i) position[i]
#define S(i) steering[i]
#define A(i) avoidance[i]
#define SP(i) speed[i]
#define LC(i) localcenter[i]
#endif


// Obstacle Data
__constant__ ObstacleData d_obstacles[MAX_OBSTACLES];
__constant__ int d_numOfObstacles;

__global__ void
steerToAvoidObstacles(VehicleData* vehicleData, VehicleConst* vehicleConst, float3 *steeringVectors)
{
    int id = (blockIdx.x * blockDim.x + threadIdx.x);
    int blockOffset = (blockDim.x * blockIdx.x * 3);
    
    // shared memory for avoidance vector
    __shared__ float3 avoidance[TPB];
    
    A(threadIdx.x) = make_float3(0, 0, 0);
    
    int intersectionFound = 0;
    int nearestIntersectionID = 0;
    float nearestIntersectionDistance = MAXFLOAT;
    
    float minDistanceToCollision = (*vehicleData).speed[id] * 5.f;
    
    int i = 0;
    
    // Find nearest obstacle
    for (; i < d_numOfObstacles; i++) {
        // find next intersection with sphere
        float b, c, d, p, q, s;
        float intersectionDistance = 0.f;
        
        float3 lc; // seems to be a lot faster if lc is local and not in shmem
        
        // find local center
        // -----------------

        lc.x = d_obstacles[i].center.x - (*vehicleData).position[id].x;
        lc.y = d_obstacles[i].center.y - (*vehicleData).position[id].y;
        lc.z = d_obstacles[i].center.z - (*vehicleData).position[id].z;
        
        
        lc = make_float3(float3Dot(lc, (*vehicleData).side[id]),
                         float3Dot(lc, (*vehicleData).up[id]),
                         float3Dot(lc, (*vehicleData).forward[id]));
        
        // compute line-sphere intersection parameters
        b = -2*lc.z;
        c = lc.x*lc.x + lc.y*lc.y + lc.z*lc.z - (d_obstacles[i].radius + (*vehicleConst).radius[id])*(d_obstacles[i].radius + (*vehicleConst).radius[id]);
        d = (b * b) - (4 * c);
        
        // path does not intersect sphere
        if (d < 0) continue;
        
        s = sqrt(d);
        p = (-b + s) / 2;
        q = (-b - s) / 2;
        
        // both intersection behind us - no potential collisions
        if ((p < 0) && (q < 0)) continue;
        
        intersectionFound = 1;
        intersectionDistance = 
            ((p > 0) && (q < 0)) ?
            // both intersections in front of us, find nearest one
            ((p < q) ? p : q) :
            // otherwise only one intersection in front of us, select it
            ((p > 0) ? p : q);
        
        if (intersectionDistance < nearestIntersectionDistance) {
            nearestIntersectionDistance = intersectionDistance;
            nearestIntersectionID = i;
        }
    }
    
    if (intersectionFound == 1 && nearestIntersectionDistance < minDistanceToCollision) {
        float3 offset = float3Sub((*vehicleData).position[id], d_obstacles[nearestIntersectionID].center);
        A(threadIdx.x) = float3PerpendicularComponent(offset, (*vehicleData).forward[id]);
        A(threadIdx.x) = float3Normalize(A(threadIdx.x));
        A(threadIdx.x) = float3Mul(A(threadIdx.x), (*vehicleConst).maxForce[id]);
        A(threadIdx.x) = float3Add(A(threadIdx.x), float3Mul((*vehicleData).forward[id], (*vehicleConst).maxForce[id] * 0.75));
    }
    
    __syncthreads();
    
    // writing back to global memory (coalesced)
    ((float*)steeringVectors)[blockOffset + threadIdx.x] = A_F(threadIdx.x);
    ((float*)steeringVectors)[blockOffset + threadIdx.x + blockDim.x] = A_F(threadIdx.x + blockDim.x);
    ((float*)steeringVectors)[blockOffset + threadIdx.x + 2*blockDim.x] = A_F(threadIdx.x + 2*blockDim.x);
}

#endif // _STEER_TO_AVOID_OBSTACLES_