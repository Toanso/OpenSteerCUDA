#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <cutil.h>
#include <stdio.h>
#include "OpenSteer/VehicleData.h"
#include "OpenSteer/PathwayData.h"
#include "OpenSteer/ObstacleData.h"
#include "CUDAKernelOptions.cu"


__global__ void
steerToAvoidObstacles(VehicleData* vehicleData, VehicleConst* vehicleConst, float3 *steeringVectors, float weight, kernel_options options);
__global__ void
steerToFollowPathKernel(VehicleData *vehicleData, float3 *steeringVectors, int *direction, float predictionTime, float weight, kernel_options options);
__global__ void
steerToStayOnPathKernel(VehicleData *vehicleData, float3 *steeringVectors, float predictionTime, float weight, kernel_options options);

__global__ void
updateKernel(VehicleData *vehicleData, VehicleConst *vehicleConst, float3 *steeringVectors, float elapsedTime, kernel_options options);

// device memory objects
static float3 *d_steeringVectors = NULL;
static VehicleData *d_vehicleData = NULL;
static VehicleConst *d_vehicleConst = NULL;
static int *d_directions = NULL;

static int first_run = 1;

void runFollowPathKernel(VehicleData *h_vehicleData, VehicleConst *h_vehicleConst, int numOfVehicles, PathwayData *h_pathwayData, int *h_directions, ObstacleData *h_obstacleData, int numOfObstacles, float elapsedTime)
{
    // init GPU
    int gpu_count;
    hipGetDeviceCount(&gpu_count);
    if (gpu_count < 1) {
        return;
    }
    
    hipSetDevice(0);
    
    dim3 grid(numOfVehicles/TPB,1,1);
    dim3 threads(TPB, 1, 1);
    
    // allocate device memory
    const unsigned int mem_size_vehicle = sizeof(VehicleData);
    if (d_vehicleData == NULL) {
        hipMalloc((void **)&d_vehicleData, mem_size_vehicle);
        hipMemcpy(d_vehicleData, h_vehicleData, mem_size_vehicle, hipMemcpyHostToDevice);
    }
    
    const unsigned int mem_size_vehicle_const = sizeof(VehicleConst);
    if (d_vehicleConst == NULL) {
        hipMalloc((void **)&d_vehicleConst, mem_size_vehicle_const);
        hipMemcpy(d_vehicleConst, h_vehicleConst, mem_size_vehicle_const, hipMemcpyHostToDevice);
    }
    
    const unsigned int mem_size_steering = sizeof(float3) * numOfVehicles;

    if (d_steeringVectors == NULL) {
        hipMalloc((void **)&d_steeringVectors, mem_size_steering);
    }
    
    hipMemset(d_steeringVectors, 0, mem_size_steering);
    
    if (d_directions == NULL) {
        const unsigned int mem_size_directions = sizeof(int) * numOfVehicles;
        hipMalloc((void **)&d_directions, mem_size_directions);
        hipMemcpy(d_directions, h_directions, mem_size_directions, hipMemcpyHostToDevice);
    }
    
    // first run initializations
    if (first_run == 1) {
        hipMemcpyToSymbol(HIP_SYMBOL("pathway"), h_pathwayData, sizeof(PathwayData), 0, hipMemcpyHostToDevice);
        //CUT_CHECK_ERROR("hipMemcpyToSymbol failed");
        hipMemcpyToSymbol(HIP_SYMBOL("d_obstacles"), h_obstacleData, sizeof(ObstacleData) * numOfObstacles, 0, hipMemcpyHostToDevice);
        hipMemcpyToSymbol(HIP_SYMBOL("d_numOfObstacles"), &numOfObstacles, sizeof(int), 0, hipMemcpyHostToDevice);
    }
    
    // start avoidObstacle kernel
    steerToAvoidObstacles<<<grid, threads>>>(d_vehicleData, d_vehicleConst, d_steeringVectors, 1.f, NONE);
    
    // start followPath kernel
    steerToFollowPathKernel<<<grid, threads>>>(d_vehicleData, d_steeringVectors, d_directions, 3.f, 1.f, IGNORE_UNLESS_ZERO);
    //steerToStayOnPathKernel<<<grid, threads>>>(d_vehicleData, d_steeringVectors, 3.f, 1.f, IGNORE_UNLESS_ZERO);
    //CUT_CHECK_ERROR("steerToFollowPathKernel execution failed");
    
    // start update kernel
    updateKernel<<<grid, threads>>>(d_vehicleData, d_vehicleConst, d_steeringVectors, elapsedTime, NONE);
    //CUT_CHECK_ERROR("updateKernel execution failed");
    
    // copy vehicle data back to host memory
    hipMemcpy(h_vehicleData, d_vehicleData, mem_size_vehicle, hipMemcpyDeviceToHost);
    
    first_run = 0;
}

void endFollowPath(void)
{
    hipFree(d_vehicleData);
    hipFree(d_vehicleConst);
    hipFree(d_steeringVectors);
    hipFree(d_directions);
    
    d_vehicleData = NULL;
    d_vehicleConst = NULL;
    d_steeringVectors = NULL;
    d_directions = NULL;
    
    first_run = 1;
}