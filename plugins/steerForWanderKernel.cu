#include "hip/hip_runtime.h"
#ifndef _STEER_FOR_WANDER_KERNEL_H_
#define _STEER_FOR_WANDER_KERNEL_H_

#include <cutil.h>
#include "VehicleData.h"
#include "WanderAroundCUDADefines.h"
#include "CUDAFloatUtilities.cu"
#include "CUDAVectorUtilities.cu"

#define CHECK_BANK_CONFLICTS 1
#if CHECK_BANK_CONFLICTS
#define S_F(i) (CUT_BANK_CHECKER(((float*)steering), i))
#define S(i) (CUT_BANK_CHECKER(steering, i))
#define SI_F(i) (CUT_BANK_CHECKER(((float*)side), i))
#define SI(i) (CUT_BANK_CHECKER(side, i))
#define U_F(i) (CUT_BANK_CHECKER(((float*)up), i))
#define U(i) (CUT_BANK_CHECKER(up, i))
#else
#define S_F(i) ((float*)steering)[i]
#define S(i) steering[i]
#define SI_F(i) ((float*)side)[i]
#define SI(i) side[i]
#define U_F(i) ((float*)up)[i]
#define U(i) up[i]
#endif

__device__ float
scalarRandomWalk(float initial, float walkspeed, float min, float max, float random);

__global__ __device__ void
steerForWander2DKernel(VehicleData *vehicleData, float *random, float dt, float3 *steeringVectors, float wanderFactor, float2 *wanderData)
{
    int id = (blockIdx.x * blockDim.x + threadIdx.x);
    int blockOffset2 = (blockDim.x * blockIdx.x);
    int blockOffset3 = (blockDim.x * blockIdx.x * 3);
    
    // shared memory for random numbers
    //__shared__ float2 random[TPB];
    
    // shared memory for side vector
    __shared__ float3 side[TPB];
    
    // shared memory for up vector
    __shared__ float3 up[TPB];
    
    // shared memory for steering vectors
    __shared__ float3 steering[TPB];
    
    // copy side vector from global memory (coalesced)
    SI_F(threadIdx.x) = ((float*)(*vehicleData).side)[blockOffset3 + threadIdx.x];
    SI_F(threadIdx.x + blockDim.x) = ((float*)(*vehicleData).side)[blockOffset3 + threadIdx.x + blockDim.x];
    SI_F(threadIdx.x + 2*blockDim.x) = ((float*)(*vehicleData).side)[blockOffset3 + threadIdx.x + 2*blockDim.x];
    
    // copy up vector from global memory (coalesced)
    U_F(threadIdx.x) = ((float*)(*vehicleData).up)[blockOffset3 + threadIdx.x];
    U_F(threadIdx.x + blockDim.x) = ((float*)(*vehicleData).up)[blockOffset3 + threadIdx.x + blockDim.x];
    U_F(threadIdx.x + 2*blockDim.x) = ((float*)(*vehicleData).up)[blockOffset3 + threadIdx.x + 2*blockDim.x];
    
    float speed = 12 * dt;
    
    float wanderSide = scalarRandomWalk(wanderData[id].x, speed, -1, +1, random[id]);
    float wanderUp = scalarRandomWalk(wanderData[id].y, speed, -1, +1, random[id+blockOffset2]);
    
    wanderData[id].x = wanderSide;
    wanderData[id].y = wanderUp;
    
    SI(threadIdx.x) = float3Mul(SI(threadIdx.x), wanderSide);
    U(threadIdx.x) = float3Mul(U(threadIdx.x), wanderUp);
    
    S(threadIdx.x).x = SI(threadIdx.x).x + U(threadIdx.x).x;
    S(threadIdx.x).y = 0.f; // SI(threadIdx.x).y + U(threadIdx.x).y;
    S(threadIdx.x).z = SI(threadIdx.x).z + U(threadIdx.x).z;
    
    // mix in wander behavior
    if (steeringVectors[id].x == 0.f && steeringVectors[id].z == 0.f) {
        
    } else {
        S(threadIdx.x) = float3BlendIn(wanderFactor, S(threadIdx.x), steeringVectors[id]);
    }


    // copy steering vector back to global memory (coalesced)
    ((float*)steeringVectors)[blockOffset3 + threadIdx.x] =  S_F(threadIdx.x);
    ((float*)steeringVectors)[blockOffset3 + threadIdx.x + blockDim.x] = S_F(threadIdx.x + blockDim.x);
    ((float*)steeringVectors)[blockOffset3 + threadIdx.x + 2*blockDim.x] = S_F(threadIdx.x + 2*blockDim.x);
}

__device__ float
scalarRandomWalk(float initial, float walkspeed, float min, float max, float random)
{
    float wander = initial + (((random * 2) - 1) * walkspeed);
    return clip(wander, min, max);
}

#endif // _STEER_FOR_FLEE_KERNEL_H_