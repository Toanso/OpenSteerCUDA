#include <hip/hip_runtime.h>

__global__ void
OneTurningCUDAKernel(int* values);

void runCUDAKernel(int* values, char* device_name)
{
    int gpu_count;
    hipGetDeviceCount(&gpu_count);
    if (gpu_count < 1) {
        return;
    }
    
    struct hipDeviceProp_t device_prop;
    hipGetDeviceProperties(&device_prop, 0);
    strcpy(device_name, device_prop.name);
    
    hipSetDevice(0);
    
    const unsigned int mem_size = sizeof(int) * 2;
    
    int* d_data;
    hipMalloc((void **) &d_data, mem_size);
    hipMemcpy(d_data, values, mem_size, hipMemcpyHostToDevice);
    
    dim3 grid(1,1,1);
    dim3 threads(1,1,1);
    
    // call kernel
    OneTurningCUDAKernel<<<grid, threads>>>(d_data);
    
    hipMemcpy(values, d_data, mem_size, hipMemcpyDeviceToHost);
    
    hipFree(d_data);
}