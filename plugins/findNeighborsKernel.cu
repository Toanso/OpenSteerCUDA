#include "hip/hip_runtime.h"
#ifndef _FIND_NEIGHBORS_KERNEL_CU_
#define _FIND_NEIGHBORS_KERNEL_CU_

#include "VehicleData.h"
#include "NeighborData.h"
#include "CUDANeighborUtilities.cu"
#include "CUDAKernelOptions.cu"
#include <stdio.h>

#define CHECK_BANK_CONFLICTS 0
#if CHECK_BANK_CONFLICTS
#define N_I(i) (CUT_BANK_CHECKER(((int*)neighbor), i))
#define F_F(i) (CUT_BANK_CHECKER(((float*)forward), i))
#define P_F(i) (CUT_BANK_CHECKER(((float*)position), i))
#define S_F(i) (CUT_BANK_CHECKER(((float*)steering), i))
#define N(i) (CUT_BANK_CHECKER(neighbor, i))
#define F(i) (CUT_BANK_CHECKER(forward, i))
#define P(i) (CUT_BANK_CHECKER(position, i))
#define S(i) (CUT_BANK_CHECKER(steering, i))
#define SP(i) (CUT_BANK_CHECKER(speed, i))
#else
#define N_I(i) ((int*)neighbor)[i]
#define F_F(i) ((float*)forward)[i]
#define P_F(i) ((float*)position)[i]
#define S_F(i) ((float*)steering)[i]
#define N(i) neighbor[i]
#define F(i) forward[i]
#define P(i) position[i]
#define S(i) steering[i]
#define SP(i) speed[i]
#endif

__global__ void
findNeighborsKernel(VehicleData* vehicleData, int* indices, int* agents, NeighborData* neighbors, float radius)
{
    int id = (blockIdx.x * blockDim.x + threadIdx.x);
    int blockOffsetNeighbors = (blockDim.x * blockIdx.x * (MAX_NEIGHBORS + 1));
    int numOfAgents = (gridDim.x * blockDim.x);
    
    // shared memort for neighbor data
    __shared__ NeighborData neighbor[TPB];
    
    // shared memory for position data
    __shared__ float3 positions[TPB][MAX_NEIGHBORS];
    
    //printf("handling id %d\n", id);
    
    // reset numbers of neighbors
    neighbor[threadIdx.x].numOfNeighbors = 0;
    int3 my_grid_cell = cellIndex((*vehicleData).position[id]);
    int cpd = floor(radius / CELL_SIZE + 1);
    
    //printf("cpd is %d\n", cpd);
    
    for (int x = -cpd; x<=cpd; ++x) {
        for (int y = -cpd; y<=cpd; ++y) {
            for (int z = -cpd; z<=cpd; ++z) {
                
                int3 offset = make_int3(x, y, z);
                int3 cell = make_int3(my_grid_cell.x + offset.x,
                                      my_grid_cell.y + offset.y,
                                      my_grid_cell.z + offset.z);
                
                //printf("x: %d, y:%d, z:%d\n", x, y, z);
             
                if (isValidCell(cell) == 0) {
                    continue;
                }
                
                // TODO: Check if cell is within search radius
                
                int cell_index = indexByCellIndex(cell);
                //printf("cell_index is %d\n", cell_index);
                int startID = indices[cell_index];
                int totalCells = (2 * WORLD_SIZE / CELL_SIZE) * (2 * WORLD_SIZE / CELL_SIZE) * (2 * WORLD_SIZE / CELL_SIZE);
                int endID = (cell_index + 1) == totalCells ? numOfAgents-1 : indices[cell_index+1];
                
                //printf("startID: %d, endID: %d, totalCells: %d\n", startID, endID, totalCells);
                
                int i = startID;
                for (; i < endID; i++) {
                    if (agents[i] != id)
                        addNeighbor(neighbor, radius, vehicleData, agents[i], id, (float3**)positions);
                }
            }
        }
    }
    
    __syncthreads();
    int i;    
    // copy neighbor data back to global memory
    for (i = 0; i < (sizeof(NeighborData) / sizeof(int)); i++) {
        ((int*)neighbors)[blockOffsetNeighbors + threadIdx.x + i*blockDim.x] = N_I(threadIdx.x + i*blockDim.x);
    }
    //neighbors[id] = neighbor[threadIdx.x];
    
    __syncthreads();
//    if (neighbors[id].numOfNeighbors > 0) 
//        printf("content is: NUM: %d VALUES (%d, %d, %d, %d, %d, %d, %d)\n", neighbors[id].numOfNeighbors, neighbors[id].idsOfNeighbors[0], neighbors[id].idsOfNeighbors[1], neighbors[id].idsOfNeighbors[2], neighbors[id].idsOfNeighbors[3], neighbors[id].idsOfNeighbors[4], neighbors[id].idsOfNeighbors[5], neighbors[id].idsOfNeighbors[6]);
    
}

#endif // _FIND_NEIGHBORS_KERNEL_CU_