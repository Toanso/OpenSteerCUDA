#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <cutil.h>
#include <stdio.h>
#include "VehicleData.h"
#include "RandomizedVector.h"
#include "MultiplePursuitCUDADefines.h"

__global__ __device__ void
steerForPursuitKernel(VehicleData *vehicleData, float3 wandererPosition, float3 wandererVelocity, float3 *steeringVectors, float maxPredictionTime);

__global__ void
updateKernel(VehicleData *vehicleData, float3 *steeringVectors, float elapsedTime);

static float3 *d_steeringVectors = NULL;
static VehicleData *d_vehicleData = NULL;
static float *d_randomVectors = NULL;
//static OpenSteer::RandomizedVector *randomizedVec = new OpenSteer::RandomizedVector(NUM_OF_AGENTS);
//static unsigned int iterations = 0;
static int first_time = 1;

void runMultiplePursuitKernel(VehicleData *h_vehicleData, float3 wandererPosition, float3 wandererVelocity, float elapsedTime, int copy_vehicle_data)
{
    const float h_timeFactorTable[9] = {2.f, 4.f, 0.85f, 2.f, 0.8f, 1.8f, 0.5f, 1.f, 4.f};
    
    int gpu_count;
    hipGetDeviceCount(&gpu_count);
    if (gpu_count < 1) {
        return;
    }
    
    hipSetDevice(0);
    
    // copy time factor table
    if (first_time == 1) {
        hipMemcpyToSymbol(HIP_SYMBOL("timeFactorTable"), h_timeFactorTable, sizeof(float) * 9, 0, hipMemcpyHostToDevice);
    }

    dim3 grid(NUM_OF_AGENTS/TPB,1,1);
    dim3 threads(TPB,1,1);
        
    // prepare memory for steeringVectors
    const unsigned int mem_size_steering = sizeof(float3) * NUM_OF_AGENTS;
    if (d_steeringVectors == NULL) {
        hipMalloc((void **) &d_steeringVectors, mem_size_steering);
    }
    
    // prepare vehicle data
    const unsigned int mem_size_vehicle = sizeof(VehicleData);
    
    if (d_vehicleData == NULL || copy_vehicle_data == 1) {
        if (d_vehicleData == NULL)
            hipMalloc((void **) &d_vehicleData, mem_size_vehicle);
        
        hipMemcpy(d_vehicleData, h_vehicleData, mem_size_vehicle, hipMemcpyHostToDevice);
    }
    
    // copy random vector
//    const unsigned int mem_size_random = sizeof(float) * randomizedVec->size();
//    if (d_randomVectors == NULL) {
//        hipMalloc((void **) &d_randomVectors, mem_size_random);
//    }
    
//    if (iterations % 20 == 0) {
//        randomizedVec->renew();
//        hipMemcpy(d_randomVectors, randomizedVec->getVector(), mem_size_random, hipMemcpyHostToDevice);
//    }
        
    // create and start timer
//    unsigned int timer = 0;
//    CUT_SAFE_CALL(cutCreateTimer(&timer));
//    CUT_SAFE_CALL(cutStartTimer(timer));
    
    // call steerForSeekKernel
    steerForPursuitKernel<<<grid, threads>>>(d_vehicleData, wandererPosition, wandererVelocity, d_steeringVectors, 20.f);
    //CUT_CHECK_ERROR("Kernel execution failed");
    
    // stop and destroy timer
//    CUT_SAFE_CALL(cutStopTimer(timer));
//    printf("Raw processing time (steerForPursuitKernel): %f (ms) \n", cutGetTimerValue(timer));
//    CUT_SAFE_CALL(cutDeleteTimer(timer));
    
    // create and start timer
//    CUT_SAFE_CALL(cutCreateTimer(&timer));
//    CUT_SAFE_CALL(cutStartTimer(timer));
    
    // call updateKernel
    updateKernel<<<grid, threads>>>(d_vehicleData, d_steeringVectors, elapsedTime);
    //CUT_CHECK_ERROR("Kernel execution failed");
    
    hipDeviceSynchronize();
    
    // stop and destroy timer
//    CUT_SAFE_CALL(cutStopTimer(timer));
//    printf("Raw processing time (updateKernel): %f (ms) \n", cutGetTimerValue(timer));
//    CUT_SAFE_CALL(cutDeleteTimer(timer));
//    CUT_SAFE_CALL(cutCreateTimer(&timer));
//    CUT_SAFE_CALL(cutStartTimer(timer));
    
    hipMemcpy(h_vehicleData, d_vehicleData, mem_size_vehicle, hipMemcpyDeviceToHost);
    
    // stop and destroy timer
//    CUT_SAFE_CALL(cutStopTimer(timer));
//    printf("Memcpy time: %f (ms) \n", cutGetTimerValue(timer));
//    CUT_SAFE_CALL(cutDeleteTimer(timer));    
    
    
    //hipFree(vehicleData);
    
    first_time = 0;
}

void endMultiplePursuit(void)
{
    hipFree(d_vehicleData);
    hipFree(d_steeringVectors);
    
    d_vehicleData = NULL;
    d_steeringVectors = NULL;
}
