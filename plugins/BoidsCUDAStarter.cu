#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "VehicleData.h"
#include "NeighborData.h"
#include "CUDAKernelOptions.cu"
#include <stdio.h>
#include <cutil.h>

__global__ void
findNeighborsKernel(VehicleData* vehicleData, int* indices, int* agents, NeighborData* neighbours, float radius);

__global__ void
steerForSeparationKernel(VehicleData *vehicleData, float3 *steeringVectors, float maxDistance, float cosMaxAngle, NeighborData* neighborData, float blendFactor, kernel_options options);

__global__ void
steerForAlignmentKernel(VehicleData *vehicleData, float3 *steeringVectors, float maxDistance, float cosMaxAngle, NeighborData* neighborData, float blendFactor, kernel_options options);

__global__ void
steerForCohesionKernel(VehicleData *vehicleData, float3 *steeringVectors, float maxDistance, float cosMaxAngle, NeighborData* neighborData, float blendFactor, kernel_options options);

__global__ void
updateKernel(VehicleData *vehicleData, float3 *steeringVectors, float elapsedTime, kernel_options options);

void debugNeighbors(VehicleData *vehicleData, NeighborData* neighborData, int numOfVehicles);



// device memory objects
static float3* d_steeringVectors = NULL;
static float3* h_steeringVectors = NULL;
static VehicleData* d_vehicleData = NULL;
static NeighborData* d_neighborData = NULL;
static NeighborData* h_neighborData = NULL;
static int* d_neighborIndices = NULL;
static int* d_neighborAgents = NULL;

void runBoidsKernel(VehicleData *h_vehicleData, int numOfVehicles, int* h_neighborIndices, int numOfNIndices, int* h_neighborAgents, int numOfNAgents, float elapsedTime)
{
    // init GPU
    int gpu_count;
    hipGetDeviceCount(&gpu_count);
    if (gpu_count < 1) {
        return;
    }
    
    hipSetDevice(0);
    
    dim3 grid(numOfVehicles/TPB, 1, 1);
    dim3 threads(TPB, 1, 1);
    
    // allocate device memory
    const unsigned int mem_size_vehicle = sizeof(VehicleData);
    if (d_vehicleData == NULL) {
        hipMalloc((void **)&d_vehicleData, mem_size_vehicle);
        hipMemcpy(d_vehicleData, h_vehicleData, mem_size_vehicle, hipMemcpyHostToDevice);
    }
    
    const unsigned int mem_size_steering = sizeof(float3) * numOfVehicles;
    
    if (d_steeringVectors == NULL) {
        hipMalloc((void **)&d_steeringVectors, mem_size_steering);
        h_steeringVectors = (float3*)malloc(mem_size_steering);
    }
    
    hipMemset(d_steeringVectors, 0, mem_size_steering);
    
    const unsigned int mem_size_neighbor_indices = sizeof(int) * numOfNIndices;
    const unsigned int mem_size_neighbor_agents = sizeof(int) * numOfNAgents;
    const unsigned int mem_size_neighbor_data = sizeof(NeighborData) * numOfVehicles;
    
    if (d_neighborData == NULL) {
        const unsigned int mem_size_neighbor_data = sizeof(NeighborData) * numOfVehicles;
        hipMalloc((void **)&d_neighborData, mem_size_neighbor_data);
        h_neighborData = (NeighborData*) malloc(mem_size_neighbor_data);
        hipMalloc((void **)&d_neighborIndices, mem_size_neighbor_indices);
        hipMalloc((void **)&d_neighborAgents, mem_size_neighbor_agents);
    }
    
    // copy neighbor raw data to global memory
    hipMemcpy(d_neighborIndices, h_neighborIndices, mem_size_neighbor_indices, hipMemcpyHostToDevice);
    hipMemcpy(d_neighborAgents, h_neighborAgents, mem_size_neighbor_agents, hipMemcpyHostToDevice);
    
    // run find neighbor kernel
    findNeighborsKernel<<<grid, threads>>>(d_vehicleData, d_neighborIndices, d_neighborAgents, d_neighborData, 4.24f);
    
    // copy neighbor data back for testing reason
    //hipMemcpy(h_neighborData, d_neighborData, mem_size_neighbor_data, hipMemcpyDeviceToHost);
    
    //debugNeighbors(h_vehicleData, h_neighborData, numOfVehicles);

    // run steer for separation kernel
    steerForSeparationKernel<<<grid, threads>>>(d_vehicleData, d_steeringVectors, 5.f, -0.707f, d_neighborData, 1.f, NONE);
    
    // run steer for alignment kernel
    steerForAlignmentKernel<<<grid, threads>>>(d_vehicleData, d_steeringVectors, 7.5f, 0.7f, d_neighborData, 0.4f, NONE);
    
    // run steer for cohesion kernel
    steerForCohesionKernel<<<grid, threads>>>(d_vehicleData, d_steeringVectors, 9.f, -0.15f, d_neighborData, 0.285f, NONE);
    
    // copy steering vectors back to test
    //hipMemcpy(h_steeringVectors, d_steeringVectors, mem_size_steering, hipMemcpyDeviceToHost);
    
//    int n;
//    for (n=0; n < numOfVehicles; n++) {
//        printf("(%d): (%f, %f, %f)\n", n, h_steeringVectors[n].x, h_steeringVectors[n].y, h_steeringVectors[n].z);
//    }
    
    // run update kernel
    updateKernel<<<grid, threads>>>(d_vehicleData, d_steeringVectors, elapsedTime, (kernel_options)(LOCAL_SPACE_BANKING | SPHERICAL_WRAP_AROUND));
    
    // copy vehicle data back to host memory
    hipMemcpy(h_vehicleData, d_vehicleData, mem_size_vehicle, hipMemcpyDeviceToHost);
}

void endBoids(void)
{
    hipFree(d_vehicleData);
    hipFree(d_steeringVectors);    
    hipFree(d_neighborIndices);
    hipFree(d_neighborAgents);
    hipFree(d_neighborData);

    d_vehicleData = NULL;
    d_steeringVectors = NULL;
    d_neighborIndices = NULL;
    d_neighborAgents = NULL;
    d_neighborData = NULL;
}

void debugNeighbors(VehicleData *vehicleData, NeighborData* neighborData, int numOfVehicles)
{
    int i;
    
    for (i = 0; i < numOfVehicles; i++) {
        float3 position = (*vehicleData).position[i];
        printf("Checking Vehicle %d: (%f, %f, %f):\n", i, position.x, position.y, position.z);
        
        int numOfNeighbors = neighborData[i].numOfNeighbors;
        printf("  Found %d neighbors:\n", numOfNeighbors);
        
        int n;
        for (n = 0; n < numOfNeighbors; n++) {
            int idOfNeighbor = neighborData[i].idsOfNeighbors[n];
            float3 positionOfNeighbor = (*vehicleData).position[idOfNeighbor];
            printf("    ID: %d Pos: (%f, %f, %f)\n", idOfNeighbor, positionOfNeighbor.x, positionOfNeighbor.y, positionOfNeighbor.z);
        }
    }
}
