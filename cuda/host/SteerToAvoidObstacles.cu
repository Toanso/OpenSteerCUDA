#include "hip/hip_runtime.h"
#include "SteerToAvoidObstacles.h"
#include <hip/hip_runtime.h>
#include "OpenSteer/VehicleData.h"
#include "OpenSteer/ObstacleData.h"
#include "CUDAKernelOptions.cu"
#include <iostream>

using namespace OpenSteer;
using namespace std;

__global__ void
steerToAvoidObstacles(VehicleData* vehicleData, VehicleConst* vehicleConst, float3 *steeringVectors, float weight, kernel_options options);

OpenSteer::SteerToAvoidObstacles::SteerToAvoidObstacles(float weight, kernel_options options)
{
    threadsPerBlock = 128;
    this->weight = weight;
    this->options = options;
}

OpenSteer::SteerToAvoidObstacles::~SteerToAvoidObstacles() {}

void OpenSteer::SteerToAvoidObstacles::init()
{
    // nothing to do
}

void OpenSteer::SteerToAvoidObstacles::run()
{
    steerToAvoidObstacles<<<gridDim(), blockDim()>>>(getVehicleData(), getVehicleConst(), getSteeringVectors(), weight, options);
}

void OpenSteer::SteerToAvoidObstacles::close()
{
    // nothing to do
}

void OpenSteer::SteerToAvoidObstacles::setObstacles(ObstacleData *obstacleData, int numOfObstacles)
{
    hipMemcpyToSymbol(HIP_SYMBOL("d_obstacles"), obstacleData, sizeof(ObstacleData) * numOfObstacles, 0, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL("d_numOfObstacles"), &numOfObstacles, sizeof(int), 0, hipMemcpyHostToDevice);
}

