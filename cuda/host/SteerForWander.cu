#include "hip/hip_runtime.h"
#include "SteerForWander.h"
#include <hip/hip_runtime.h>
#include "OpenSteer/VehicleData.h"
#include "CUDAKernelOptions.cu"
#include <iostream>

using namespace OpenSteer;
using namespace std;

__global__ void
steerForWander2DKernel(VehicleData *vehicleData, float *random, float dt, float3 *steeringVectors, float2 *wanderData, float weight, kernel_options options);

OpenSteer::SteerForWander::SteerForWander(float dt, float weight, kernel_options options)
{
    d_randomNumbers = NULL;
    d_wanderData = NULL;
    randomizedVector = NULL;
    threadsPerBlock = 128;
    this->dt = dt;
    this->weight = weight;
    this->options = options;
}

OpenSteer::SteerForWander::~SteerForWander() {}

void OpenSteer::SteerForWander::init()
{
    // random number generator
    randomizedVector = new RandomizedVector(2*getNumberOfAgents());
    
    // device memory for wander data
    mem_size_wander = getNumberOfAgents()*sizeof(float2);
    hipError_t retval = hipMalloc((void **)&d_wanderData, mem_size_wander);
    if (retval != hipSuccess)
        cout << "Error while allocating d_wanderData memory: " << hipGetErrorString(retval) << endl;
    
    // device memory for random numbers
    mem_size_random = randomizedVector->size() * sizeof(float);
    retval = hipMalloc((void **)&d_randomNumbers, mem_size_random);
    if (retval != hipSuccess)
        cout << "Error while allocating d_randomNumbers memory: " << hipGetErrorString(retval) << endl;
        
}

void OpenSteer::SteerForWander::run()
{
    steerForWander2DKernel<<<gridDim(), blockDim()>>>(getVehicleData(), d_randomNumbers, dt, getSteeringVectors(), d_wanderData, weight, options);
}

void OpenSteer::SteerForWander::close()
{
    if (d_wanderData != NULL) {
        hipFree(d_wanderData);
        d_wanderData = NULL;        
    }
    
    if (d_randomNumbers != NULL) {
        hipFree(d_randomNumbers);
        d_randomNumbers = NULL;
    }
    
    if (randomizedVector != NULL) {
        delete randomizedVector;
        randomizedVector = NULL;        
    }
}
