#include "hip/hip_runtime.h"
#include "SteerForCohesion.h"
#include <hip/hip_runtime.h>
#include "OpenSteer/VehicleData.h"
#include "OpenSteer/NeighborDataProvider.h"
#include "CUDAKernelOptions.cu"
#include <iostream>

using namespace OpenSteer;
using namespace std;

__global__ void
steerForCohesionKernel(VehicleData *vehicleData, VehicleConst *vehicleConst, float3 *steeringVectors, float maxDistance, float cosMaxAngle, NeighborData* neighborData, float weight, kernel_options options);

OpenSteer::SteerForCohesion::SteerForCohesion(NeighborDataProvider* neighborDataProvider, float maxDistance, float cosMaxAngle, float weight, kernel_options options)
{
    threadsPerBlock = 128;
    this->weight = weight;
    this->options = options;
    this->neighborDataProvider = neighborDataProvider;
    this->maxDistance = maxDistance;
    this->cosMaxAngle = cosMaxAngle;
}

OpenSteer::SteerForCohesion::~SteerForCohesion() {}

void OpenSteer::SteerForCohesion::init()
{
    // nothing to do
}

void OpenSteer::SteerForCohesion::run()
{
    steerForCohesionKernel<<<gridDim(), blockDim()>>>(getVehicleData(), getVehicleConst(), getSteeringVectors(), maxDistance, cosMaxAngle, neighborDataProvider->getNeighborData(), weight, options);
}

void OpenSteer::SteerForCohesion::close()
{
    // nothing to do
}
