#include "hip/hip_runtime.h"
#include "FindFollower.h"
#include <hip/hip_runtime.h>
#include "OpenSteer/VehicleData.h"
#include "CUDAKernelOptions.cu"
#include <iostream>

using namespace OpenSteer;
using namespace std;

__global__ void
findFollowerKernel(VehicleData *vehicleData, float3 *seekVectors);

OpenSteer::FindFollower::FindFollower()
{
    d_seekVectors = NULL;
    threadsPerBlock = 128;
}

OpenSteer::FindFollower::~FindFollower() {}

void OpenSteer::FindFollower::init()
{
    // device memory for seek vector
    mem_size_seek_vectors = getNumberOfAgents()*sizeof(float3);
    hipError_t retval = hipMalloc((void **)&d_seekVectors, mem_size_seek_vectors);
    if (retval != hipSuccess)
        cout << "Error while allocating d_seekVectors memory: " << hipGetErrorString(retval) << endl;
}

void OpenSteer::FindFollower::run()
{    
    findFollowerKernel<<<gridDim(), blockDim()>>>(getVehicleData(), d_seekVectors);
}

void OpenSteer::FindFollower::close()
{
    if (d_seekVectors != NULL) {
        hipFree(d_seekVectors);
        d_seekVectors = NULL;        
    }
}
