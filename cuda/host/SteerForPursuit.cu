#include "hip/hip_runtime.h"
#include "SteerForPursuit.h"
#include <hip/hip_runtime.h>
#include "OpenSteer/VehicleData.h"
#include "OpenSteer/PursuitDataProvider.h"
#include "CUDAKernelOptions.cu"
#include <iostream>

using namespace OpenSteer;
using namespace std;

__global__ void
steerForPursuitKernel(VehicleData *vehicleData, float3 *quarryPosition, float3 *quarryVelocity, float3 *steeringVectors, float maxPredictionTime, float weight, kernel_options options);

OpenSteer::SteerForPursuit::SteerForPursuit(PursuitDataProvider* pursuitDataProvider, float maxPredictionTime, float weight, kernel_options options)
{
    threadsPerBlock = 128;
    this->weight = weight;
    this->options = options;
    this->pursuitDataProvider = pursuitDataProvider;
    this->maxPredictionTime = maxPredictionTime;
}

OpenSteer::SteerForPursuit::~SteerForPursuit() {}

void OpenSteer::SteerForPursuit::init()
{
    const float h_timeFactorTable[9] = {2.f, 4.f, 0.85f, 2.f, 0.8f, 1.8f, 0.5f, 1.f, 4.f};
    hipMemcpyToSymbol(HIP_SYMBOL("timeFactorTable"), h_timeFactorTable, sizeof(float) * 9, 0, hipMemcpyHostToDevice);
}

void OpenSteer::SteerForPursuit::run()
{
    steerForPursuitKernel<<<gridDim(), blockDim()>>>(getVehicleData(), pursuitDataProvider->getPursuitPosition(), pursuitDataProvider->getPursuitVelocity(), getSteeringVectors(), maxPredictionTime, weight, options);
}

void OpenSteer::SteerForPursuit::close()
{
    // nothing to do
}
