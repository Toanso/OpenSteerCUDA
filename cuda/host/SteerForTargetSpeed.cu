#include "hip/hip_runtime.h"
#include "SteerForTargetSpeed.h"
#include <hip/hip_runtime.h>
#include "OpenSteer/VehicleData.h"
#include "OpenSteer/SeekVectorProvider.h"
#include "CUDAKernelOptions.cu"
#include <iostream>

using namespace OpenSteer;
using namespace std;

__global__ void
steerForTargetSpeedKernel(VehicleData *vehicleData, VehicleConst *vehicleConst, float *targetSpeeds, float3 *steeringVectors, float weight, kernel_options options);

OpenSteer::SteerForTargetSpeed::SteerForTargetSpeed(TargetSpeedProvider* targetSpeedProvider, float weight, kernel_options options)
{
    threadsPerBlock = 128;
    this->targetSpeedProvider = targetSpeedProvider;
    this->weight = weight;
    this->options = options;
}

OpenSteer::SteerForTargetSpeed::~SteerForTargetSpeed() {}

void OpenSteer::SteerForTargetSpeed::init()
{
    // nothing to do
}

void OpenSteer::SteerForTargetSpeed::run()
{
    steerForTargetSpeedKernel<<<gridDim(), blockDim()>>>(getVehicleData(), getVehicleConst(), targetSpeedProvider->getTargetSpeeds(), getSteeringVectors(), weight, options);
}

void OpenSteer::SteerForTargetSpeed::close()
{
    // nothing to do
}
